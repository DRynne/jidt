#include "gpuKnnBF_kernel.cu"
#include <stdio.h>
#include "helperfunctions.cu"
#include "ctimer.h"

#ifdef __cplusplus
extern "C" {
#endif
int cudaFindKnn(int* h_bf_indexes, float* h_bf_distances, float* h_pointset,
    float* h_query, int kth, int thelier, int nchunks, int pointdim,
    int signallength, unsigned int useMaxNorm) {
  float *d_bf_pointset, *d_bf_query;
  int *d_bf_indexes;
  float *d_bf_distances;

  unsigned int meminputsignalquerypointset= pointdim * signallength * sizeof(float);
  unsigned int mem_bfcl_outputsignaldistances= kth * signallength * sizeof(float);
  unsigned int mem_bfcl_outputsignalindexes = kth * signallength * sizeof(int);

  CPerfTimer pt1 = startTimer("kNN allocate and upload");
  checkCudaErrors( hipMalloc( (void**) &(d_bf_query), meminputsignalquerypointset));
  checkCudaErrors( hipMalloc( (void**) &(d_bf_pointset), meminputsignalquerypointset));
  //GPU output
  checkCudaErrors( hipMalloc( (void**) &(d_bf_distances), mem_bfcl_outputsignaldistances ));
  checkCudaErrors( hipMalloc( (void**) &(d_bf_indexes), mem_bfcl_outputsignalindexes ));

  hipError_t error = hipGetLastError();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }

  //Upload input data
  checkCudaErrors( hipMemcpy(d_bf_query, h_query, meminputsignalquerypointset, hipMemcpyHostToDevice ));
  checkCudaErrors( hipMemcpy(d_bf_pointset, h_pointset, meminputsignalquerypointset, hipMemcpyHostToDevice ));
  error = hipGetLastError();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }
  stopTimer(pt1);
  CPerfTimer pt2 = startTimer("kNN kernel");
  // Kernel parameters
  dim3 threads(1,1,1);
  dim3 grid(1,1,1);
  threads.x = 512;
  grid.x = (signallength-1)/threads.x + 1;
  int memkernel = kth*sizeof(float)*threads.x+\
          kth*sizeof(int)*threads.x;
  int triallength = signallength / nchunks;

  // Pointer to the function used to calculate norms
  normFunction_t *normFunction;
  hipMalloc( (void **) &normFunction, sizeof(normFunction_t) );
  if (useMaxNorm) {
    hipMemcpyFromSymbol(normFunction, HIP_SYMBOL(pMaxNorm), sizeof(normFunction_t));
  } else {
    hipMemcpyFromSymbol(normFunction, HIP_SYMBOL(pSquareNorm), sizeof(normFunction_t));
  }

  // Launch kernel
  kernelKNNshared<<<grid.x, threads.x, memkernel>>>(d_bf_query, d_bf_pointset,
      d_bf_indexes, d_bf_distances, pointdim, triallength, signallength, kth,
      thelier, normFunction);

  checkCudaErrors( hipDeviceSynchronize() );
  stopTimer(pt2);
  CPerfTimer pt3 = startTimer("kNN download and free");

  //Download result
  checkCudaErrors( hipMemcpy( h_bf_distances, d_bf_distances, mem_bfcl_outputsignaldistances, hipMemcpyDeviceToHost) );
  checkCudaErrors( hipMemcpy( h_bf_indexes, d_bf_indexes, mem_bfcl_outputsignalindexes, hipMemcpyDeviceToHost) );
  error = hipGetLastError();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }

  //Free resources
  checkCudaErrors(hipFree(d_bf_query));
  checkCudaErrors(hipFree(d_bf_pointset));
  checkCudaErrors(hipFree(d_bf_distances));
  checkCudaErrors(hipFree(d_bf_indexes));
  hipFree(normFunction);
  stopTimer(pt3);
  // hipDeviceReset();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }
  
  return 1;
}
#ifdef __cplusplus
}
#endif

#ifdef __cplusplus
extern "C" {
#endif
int cudaFindKnnSetGPU(int* h_bf_indexes, float* h_bf_distances,
    float* h_pointset, float* h_query, int kth, int thelier, int nchunks,
    int pointdim, int signallength, unsigned int useMaxNorm, int deviceid) {
  hipSetDevice(deviceid);
  return cudaFindKnn(h_bf_indexes, h_bf_distances, h_pointset, h_query,
      kth, thelier, nchunks, pointdim, signallength, useMaxNorm);
}
#ifdef __cplusplus
}
#endif

/*
 * Range search being radius a vector of length number points in queryset/pointset
 */
#ifdef __cplusplus
extern "C" {
#endif
int cudaFindRSAll(int* h_bf_npointsrange, float* h_pointset, float* h_query,
    float* h_vecradius, int thelier, int nchunks, int pointdim,
    int signallength, unsigned int useMaxNorm) {

  float *d_bf_pointset, *d_bf_query, *d_bf_vecradius;
  int *d_bf_npointsrange;

  unsigned int meminputsignalquerypointset= pointdim * signallength * sizeof(float);
  unsigned int mem_bfcl_outputsignalnpointsrange= signallength * sizeof(int);
  unsigned int mem_bfcl_inputvecradius = signallength * sizeof(float);

  CPerfTimer pt1 = startTimer("RS allocate and upload");
  checkCudaErrors( hipMalloc( (void**) &(d_bf_query), meminputsignalquerypointset));
  checkCudaErrors( hipMalloc( (void**) &(d_bf_pointset), meminputsignalquerypointset));
  checkCudaErrors( hipMalloc( (void**) &(d_bf_npointsrange), mem_bfcl_outputsignalnpointsrange ));
    checkCudaErrors( hipMalloc( (void**) &(d_bf_vecradius), mem_bfcl_inputvecradius ));

    hipError_t error = hipGetLastError();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }
  //Upload input data
  checkCudaErrors( hipMemcpy(d_bf_query, h_query, meminputsignalquerypointset, hipMemcpyHostToDevice ));
  checkCudaErrors( hipMemcpy(d_bf_pointset, h_pointset, meminputsignalquerypointset, hipMemcpyHostToDevice ));
    checkCudaErrors( hipMemcpy(d_bf_vecradius, h_vecradius, mem_bfcl_inputvecradius, hipMemcpyHostToDevice ));

    error = hipGetLastError();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }
  stopTimer(pt1);
  CPerfTimer pt2 = startTimer("RS kernel");

  // Kernel parameters
  dim3 threads(1,1,1);
  dim3 grid(1,1,1);
  threads.x = 512;
  grid.x = (signallength-1)/threads.x + 1;
  int memkernel = sizeof(int)*threads.x;
  int triallength = signallength / nchunks;

  // Pointer to the function used to calculate norms
  normFunction_t *normFunction;
  hipMalloc( (void **) &normFunction, sizeof(normFunction_t) );
  if (useMaxNorm) {
    hipMemcpyFromSymbol(normFunction, HIP_SYMBOL(pMaxNorm), sizeof(normFunction_t));
  } else {
    hipMemcpyFromSymbol(normFunction, HIP_SYMBOL(pSquareNorm), sizeof(normFunction_t));
  }

  // Launch kernel
  kernelBFRSAllshared<<< grid.x, threads.x, memkernel>>>(
          d_bf_query, d_bf_pointset, d_bf_npointsrange, pointdim,
          triallength, signallength, thelier, d_bf_vecradius, normFunction);

  checkCudaErrors(hipDeviceSynchronize());
  stopTimer(pt2);
  CPerfTimer pt3 = startTimer("RS download and free");

  checkCudaErrors( hipMemcpy( h_bf_npointsrange, d_bf_npointsrange,mem_bfcl_outputsignalnpointsrange, hipMemcpyDeviceToHost) );


  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }

  // Free resources
  checkCudaErrors(hipFree(d_bf_query));
  checkCudaErrors(hipFree(d_bf_pointset));
  checkCudaErrors(hipFree(d_bf_npointsrange));
  checkCudaErrors(hipFree(d_bf_vecradius));
  checkCudaErrors(hipFree(normFunction));
  hipDeviceReset();
  if(error!=hipSuccess){
    fprintf(stderr,"%s",hipGetErrorString(error));
    return 0;
  }
  stopTimer(pt3);
  
  return 1;
}
#ifdef __cplusplus
}
#endif

#ifdef __cplusplus
extern "C" {
#endif
int cudaFindRSAllSetGPU(int* h_bf_npointsrange, float* h_pointset,
    float* h_query, float* h_vecradius, int thelier, int nchunks,
    int pointdim, int signallength, unsigned int useMaxNorm, int deviceid) {
  hipSetDevice(deviceid);
  return cudaFindRSAll(h_bf_npointsrange, h_pointset, h_query, h_vecradius,
      thelier, nchunks, pointdim, signallength, useMaxNorm);
}
#ifdef __cplusplus
}
#endif


#ifdef __cplusplus
extern "C" {
#endif
int findRadiiAlgorithm2(float *radii, const float *data, const int *indexes,
    unsigned int k, unsigned int dim, unsigned int N) {

  unsigned int i, j;

  for (j = 0; j < N; j++) {
    radii[j] = 0.0f;
    for (i = 0; i < k; i++) {
      float d = maxMetricPoints(data + j, data + indexes[j + i*N], dim, N);
      if (d > radii[j]) {
        radii[j] = d;
      }
    }
  }

  return 1;

}
#ifdef __cplusplus
}
#endif



#ifdef __cplusplus
extern "C" {
#endif
int computeSumDigammas(float *sumDiGammas, int *nx, int *ny, unsigned int N) {

  int *d_nx, *d_ny;
  float *d_sumDiGammas, *partialSumDiGammas;

  unsigned int threads_per_block = 512;
  dim3 n_blocks, n_threads;
  n_blocks.x = ((N % threads_per_block) != 0) ? (N / threads_per_block + 1) : (N / threads_per_block);
  n_threads.x = threads_per_block;
  int memkernel = sizeof(int)*n_threads.x;

  partialSumDiGammas = (float *) malloc(n_blocks.x * sizeof(float));

  checkCudaErrors( hipMalloc((void **) &d_nx, N * sizeof(int)) );
  checkCudaErrors( hipMalloc((void **) &d_ny, N * sizeof(int)) );
  checkCudaErrors( hipMalloc((void **) &d_sumDiGammas, n_blocks.x * sizeof(float)) );

  checkCudaErrors( hipMemcpy(d_nx, nx, N*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_ny, ny, N*sizeof(int), hipMemcpyHostToDevice) );

  // printf("blocks = %i, threads = %i\n", n_blocks.x, n_threads.x);

  // reduce6<<<n_blocks, n_threads>>>(d_nx, d_ny, d_sumDiGammas, N);
  reduce1<<<n_blocks, n_threads, memkernel>>>(d_nx, d_ny, d_sumDiGammas, N);

  checkCudaErrors( hipDeviceSynchronize() );

  checkCudaErrors( hipMemcpy(partialSumDiGammas, d_sumDiGammas, n_blocks.x * sizeof(float), hipMemcpyDeviceToHost) );

  checkCudaErrors( hipDeviceSynchronize() );

  float tmp = 0;
  for (int i = 0; i < n_blocks.x; i++) {
    // printf("From block %d we got %f\n", i, partialSumDiGammas[i]);
    tmp += partialSumDiGammas[i];
  }
  *sumDiGammas = tmp;

  free(partialSumDiGammas);
  hipFree(d_nx);
  hipFree(d_ny);
  hipFree(d_sumDiGammas);

  return 1;

}
#ifdef __cplusplus
}
#endif


#ifdef __cplusplus
extern "C" {
#endif
int parallelDigammas(float *digammas, int *nx, int *ny, int signallength) {

  int *d_nx, *d_ny;
  float *d_digammas;

  // Kernel parameters
  dim3 threads(1,1,1);
  dim3 grid(1,1,1);
  threads.x = 512;
  grid.x = (signallength-1)/threads.x + 1;

  checkCudaErrors( hipMalloc((void **) &d_nx, signallength * sizeof(int)) );
  checkCudaErrors( hipMalloc((void **) &d_ny, signallength * sizeof(int)) );
  checkCudaErrors( hipMalloc((void **) &d_digammas, signallength * sizeof(float)) );

  checkCudaErrors( hipMemcpy(d_nx, nx, signallength*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_ny, ny, signallength*sizeof(int), hipMemcpyHostToDevice) );

  // printf("blocks = %i, threads = %i\n", n_blocks.x, n_threads.x);

  // Launch kernel
  gpuDigammas<<<grid.x, threads.x>>>(d_digammas, d_nx, d_ny, signallength);

  checkCudaErrors( hipDeviceSynchronize() );

  checkCudaErrors( hipMemcpy(digammas, d_digammas, signallength * sizeof(float), hipMemcpyDeviceToHost) );

  checkCudaErrors( hipDeviceSynchronize() );

  checkCudaErrors( hipFree(d_nx) );
  checkCudaErrors( hipFree(d_ny) );
  checkCudaErrors( hipFree(d_digammas) );

  return 1;
}
#ifdef __cplusplus
}
#endif


#ifdef __cplusplus
extern "C" {
#endif
void device_reset(void) {
  hipDeviceReset();
}
#ifdef __cplusplus
}
#endif

#ifdef __cplusplus
extern "C" {
#endif
void gpuWarmUp(void) {
  hipSetDevice(0);
}
#ifdef __cplusplus
}
#endif

